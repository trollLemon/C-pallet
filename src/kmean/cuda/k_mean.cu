#include "hip/hip_runtime.h"
#include "adv_color.h"
#include "cuda_cluster.h"
#include "k_mean_cuda.h"
#include "util.h"
#include <algorithm>
#include <assert.h>
#include <cmath>
#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <functional>
#include <iostream>
#include <random>
#include <unordered_set>
#define THREADS_PER_BLOCK 256
typedef pixel CUDA_COLOR_DATA;

__device__ double euclideanDistance(CUDA_COLOR_DATA x, CUDA_COLOR_DATA y) {
  double dl = static_cast<double>(y.r) - static_cast<double>(x.r);
  double da = static_cast<double>(y.g) - static_cast<double>(x.g);
  double db = static_cast<double>(y.b) - static_cast<double>(x.b);
  return sqrt(dl * dl + da * da + db * db);
}

__global__ void sumClusters(CUDA_COLOR_DATA *d_centroids,
                            CUDA_COLOR_DATA *d_colors, int *assignments,
                            int *d_clusterSizes, CUDA_COLOR_DATA *partialSums,
                            int k, int color_count) {

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  if (id > color_count)
    return;

  __shared__ CUDA_COLOR_DATA s_datapoints[THREADS_PER_BLOCK];
  s_datapoints[tid] = d_colors[id];

  __shared__ int s_clust_assn[THREADS_PER_BLOCK];
  s_clust_assn[tid] = assignments[id];

  /* int clusterId = assignments[id];
  int r, g, b;

  r = d_colors[id].r;
  g = d_colors[id].g;
  b = d_colors[id].b;

  atomicAdd(&d_clusterSizes[clusterId], 1);
  atomicAdd(&d_centroids[clusterId].r, r);
  atomicAdd(&d_centroids[clusterId].g, g);
  atomicAdd(&d_centroids[clusterId].b, b);
*/

  __syncthreads();

  if (tid == 0) {

    CUDA_COLOR_DATA b_clust_datapoint_sums[THREADS_PER_BLOCK] = {0};
    for (int j = 0; j < blockDim.x; ++j) {
      int clusterId = s_clust_assn[j];
      int r, g, b;

      r = s_datapoints[id].r;
      g = s_datapoints[id].g;
      b = s_datapoints[id].b;

      b_clust_datapoint_sums[clusterId].r += r;
      b_clust_datapoint_sums[clusterId].g += g;
      b_clust_datapoint_sums[clusterId].b += b;
    }

    for (int z = 0; z < k; ++z) {

      atomicAdd(&d_centroids[z].r, b_clust_datapoint_sums[z].r);
      atomicAdd(&d_centroids[z].g, b_clust_datapoint_sums[z].g);
      atomicAdd(&d_centroids[z].b, b_clust_datapoint_sums[z].b);
    }
    __syncthreads();
  }
}

__global__ void recalcClusters(CUDA_COLOR_DATA *d_centroids, int *d_clust_sizes,
                               int k) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx > k)
    return;

  int size = d_clust_sizes[idx];

  d_centroids[idx].r /= size;
  d_centroids[idx].g /= size;
  d_centroids[idx].b /= size;
}

__global__ void assignPoints(CUDA_COLOR_DATA *d_clusters,
                             CUDA_COLOR_DATA *d_colors, int *assignments,
                             int *cluster_counts, int k, int color_count) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx > color_count)
    return;

  double min_dist = INFINITY;
  int closest_centroid = 0;

  for (int i = 0; i < k; ++i) {
    double dist = euclideanDistance(d_clusters[i], d_colors[idx]);

    if (dist < min_dist) {
      min_dist = dist;
      closest_centroid = i;
    }
  }
  assignments[idx] = closest_centroid;
}

__global__ void initClusters(CUDA_COLOR_DATA *d_clusters,
                             CUDA_COLOR_DATA *d_colors, int size,
                             int *cluster_picks) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < size) {
    d_clusters[idx] = d_colors[cluster_picks[idx]];
  }
}

std::vector<std::string> CudaKmeanWrapper(CUDA_COLOR_DATA *pixel_data, int size,
                                          int totalPixels) {

  CUDA_COLOR_DATA *d_colors;
  CUDA_COLOR_DATA *d_clusters;
  CUDA_COLOR_DATA *d_partialSums;

  int *d_assignments;
  int *d_random_points;
  int *d_clust_sizes;

  hipMalloc((void **)&d_colors, totalPixels * sizeof(CUDA_COLOR_DATA));
  hipMalloc((void **)&d_partialSums, totalPixels * sizeof(CUDA_COLOR_DATA));
  hipMalloc((void **)&d_random_points, totalPixels * sizeof(CUDA_COLOR_DATA));
  hipMalloc((void **)&d_assignments, totalPixels * sizeof(int));
  hipMalloc((void **)&d_clusters, size * sizeof(CUDA_COLOR_DATA));
  hipMalloc(&d_clust_sizes, size * sizeof(int));
  hipMemcpy(d_colors, pixel_data, totalPixels * sizeof(CUDA_COLOR_DATA),
             hipMemcpyHostToDevice);

  int blocksPerGrid = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  std::random_device rd;
  std::seed_seq ss{rd(), rd(), rd(), rd(), rd(), rd(), rd(), rd()};

  std::mt19937 mt{ss};
  std::uniform_int_distribution<> kPoints{totalPixels / 2, totalPixels};
  std::set<int> seen; // make sure we have unique numbers
  int *colorIndecies = new int[size];
  int colorIndecies_idx = 0;
  while (colorIndecies_idx != size) {
    int num = kPoints(mt);
    if (seen.count(num) == 0) {
      seen.insert(num);
      colorIndecies[colorIndecies_idx] = num;
      colorIndecies_idx++;
    }
  }

  hipMemcpy(d_random_points, colorIndecies,
             colorIndecies_idx * sizeof(CUDA_COLOR_DATA),
             hipMemcpyHostToDevice);

  initClusters<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_clusters, d_colors, size,
                                                     d_random_points);

  hipDeviceSynchronize();

  blocksPerGrid = (totalPixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  int x = 0;
  while (x != 10) {

    assignPoints<<<blocksPerGrid, THREADS_PER_BLOCK>>>(
        d_clusters, d_colors, d_assignments, d_clust_sizes, size, totalPixels);

    sumClusters<<<blocksPerGrid, THREADS_PER_BLOCK,
                  size * sizeof(CUDA_COLOR_DATA)>>>(
        d_clusters, d_colors, d_assignments, d_clust_sizes, d_partialSums, size,
        totalPixels);

    recalcClusters<<<size, 1>>>(d_clusters, d_clust_sizes, size);
    hipMemset(d_clust_sizes, 0, size * sizeof(int));
    hipDeviceSynchronize();
    x++;
  }

  CUDA_COLOR_DATA *h_colors =
      (CUDA_COLOR_DATA *)malloc(size * sizeof(CUDA_COLOR_DATA));

  hipMemcpy(h_colors, d_clusters, size * sizeof(CUDA_COLOR_DATA),
             hipMemcpyDeviceToHost);

  hipFree(d_colors);
  hipFree(d_random_points);
  hipFree(d_assignments);
  hipFree(d_clusters);

  ADV_Color color_helper(0, 0, 0);
  std::vector<std::string> palette;
  for (int i = 0; i < size; ++i) {

    CUDA_COLOR_DATA color = h_colors[i];

    color_helper.setRGB(color.r, color.g, color.b);

    palette.push_back(color_helper.asHex());
  }

  free(h_colors);

  return palette;
}
