#include "hip/hip_runtime.h"
#include "adv_color.h"
#include "cuda_cluster.h"
#include "k_mean_cuda.h"
#include "util.h"
#include <algorithm>
#include <assert.h>
#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <functional>
#include <iostream>
#include <random>
#include <unordered_set>
#define THREADS_PER_BLOCK 256
typedef pixel CUDA_COLOR_DATA;

__device__ double euclideanDistance(CUDA_COLOR_DATA x, CUDA_COLOR_DATA y) {
  double dl = static_cast<double>(y.r) - static_cast<double>(x.r);
  double da = static_cast<double>(y.g) - static_cast<double>(x.g);
  double db = static_cast<double>(y.b) - static_cast<double>(x.b);
  return sqrt(dl * dl + da * da + db * db);
}

__global__ void recalcClusters(CUDA_COLOR_DATA *d_clusters,
                               CUDA_COLOR_DATA *d_colors, int *assignments,
                               int k, int color_count) {

  int cluster = blockIdx.x;
  int tid = threadIdx.x;

  __shared__ int sumR_shared[THREADS_PER_BLOCK];
  __shared__ int sumG_shared[THREADS_PER_BLOCK];
  __shared__ int sumB_shared[THREADS_PER_BLOCK];

  int sumR = 0;
  int sumG = 0;
  int sumB = 0;
  int count = 0;

  for (int i = tid; i < color_count; i += blockDim.x) {
    if (assignments[i] == cluster) {
      sumR += d_colors[i].r;
      sumG += d_colors[i].g;
      sumB += d_colors[i].b;
      count++;
    }
  }

  sumR_shared[tid] = sumR;
  sumG_shared[tid] = sumG;
  sumB_shared[tid] = sumB;

  __syncthreads(); 

  if (tid == 0) {
    for (int i = 1; i < blockDim.x; i++) {
      sumR += sumR_shared[i];
      sumG += sumG_shared[i];
      sumB += sumB_shared[i];
    }

    if (count > 0) {
      d_clusters[cluster].r = sumR / count;
      d_clusters[cluster].g = sumG / count;
      d_clusters[cluster].b = sumB / count;
    }
  }
}

__global__ void assignPoints(CUDA_COLOR_DATA *d_clusters,
                             CUDA_COLOR_DATA *d_colors, int *assignments, int k,
                             int color_count) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx > color_count)
    return;

  double min_dist = INFINITY;
  int closest_centroid = 0;

  for (int i = 0; i < k; ++i) {
    double dist = euclideanDistance(d_clusters[i], d_colors[idx]);

    if (dist < min_dist) {
      min_dist = dist;
      closest_centroid = i;
    }
  }
  assignments[idx] = closest_centroid;
}

__global__ void initClusters(CUDA_COLOR_DATA *d_clusters,
                             CUDA_COLOR_DATA *d_colors, int size,
                             int *cluster_picks) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < size) {
    d_clusters[idx] = d_colors[cluster_picks[idx]];
  }
}

std::vector<std::string> CudaKmeanWrapper(CUDA_COLOR_DATA *pixel_data, int size,
                                          int totalPixels) {

  CUDA_COLOR_DATA *d_colors;
  CUDA_COLOR_DATA *d_clusters;
  int *d_assignments;
  int *d_random_points;

  hipMalloc((void **)&d_colors, totalPixels * sizeof(CUDA_COLOR_DATA));
  hipMalloc((void **)&d_random_points, totalPixels * sizeof(CUDA_COLOR_DATA));
  hipMalloc((void **)&d_assignments, totalPixels * sizeof(int));
  hipMalloc((void **)&d_clusters, size * sizeof(CUDA_COLOR_DATA));

  hipMemcpy(d_colors, pixel_data, totalPixels * sizeof(CUDA_COLOR_DATA),
             hipMemcpyHostToDevice);

  int blocksPerGrid = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  std::random_device rd;
  std::seed_seq ss{rd(), rd(), rd(), rd(), rd(), rd(), rd(), rd()};

  std::mt19937 mt{ss};
  std::uniform_int_distribution<> kPoints{totalPixels / 2, totalPixels};
  std::set<int> seen; // make sure we have unique numbers
  int *colorIndecies = new int[size];
  int colorIndecies_idx = 0;
  while (colorIndecies_idx != size) {
    int num = kPoints(mt);
    if (seen.count(num) == 0) {
      seen.insert(num);
      colorIndecies[colorIndecies_idx] = num;
      colorIndecies_idx++;
    }
  }

  hipMemcpy(d_random_points, colorIndecies,
             colorIndecies_idx * sizeof(CUDA_COLOR_DATA),
             hipMemcpyHostToDevice);

  initClusters<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_clusters, d_colors, size,
                                                     d_random_points);
  hipDeviceSynchronize();

  int x = 0;
  while (x++ != 5) {

    blocksPerGrid = (totalPixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    assignPoints<<<blocksPerGrid, THREADS_PER_BLOCK>>>(
        d_clusters, d_colors, d_assignments, size, totalPixels);
    recalcClusters<<<size, 1>>>(d_clusters, d_colors, d_assignments, size,
                                totalPixels);

    hipDeviceSynchronize();
  }

  CUDA_COLOR_DATA *h_colors =
      (CUDA_COLOR_DATA *)malloc(size * sizeof(CUDA_COLOR_DATA));

  hipMemcpy(h_colors, d_clusters, size * sizeof(CUDA_COLOR_DATA),
             hipMemcpyDeviceToHost);

  ADV_Color color_helper(0, 0, 0);
  std::vector<std::string> palette;
  for (int i = 0; i < size; ++i) {

    CUDA_COLOR_DATA color = h_colors[i];

    color_helper.setRGB(color.r, color.g, color.b);

    palette.push_back(color_helper.asHex());
  }

  return palette;
}
